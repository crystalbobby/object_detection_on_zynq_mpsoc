
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"


__global__ void quantize(int N, float scale, float *tensor)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<N) tensor[i] = round(scale*tensor[i])/scale;
}

__global__ void quantize2(int N, float scale, float *tensor, float *output)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<N) output[i] = round(scale*tensor[i])/scale;
}



void doquant(int N, float scale, float *tensor){

quantize<<<(N+255)/256, 256>>>(N, scale, tensor);

}

void doquant2(int N, float scale, float *tensor, float *output){

quantize2<<<(N+255)/256, 256>>>(N, scale, tensor, output);

}

#endif
